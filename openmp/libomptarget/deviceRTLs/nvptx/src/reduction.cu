#include "hip/hip_runtime.h"
//===---- reduction.cu - NVPTX OpenMP reduction implementation ---- CUDA
//-*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of reduction with KMPC interface.
//
//===----------------------------------------------------------------------===//

#include <complex.h>
#include <stdio.h>

#include "omptarget-nvptx.h"

EXTERN
void __kmpc_nvptx_end_reduce(int32_t global_tid) {}

EXTERN
void __kmpc_nvptx_end_reduce_nowait(int32_t global_tid) {}

EXTERN int32_t __kmpc_shuffle_int32(int32_t val, int16_t delta, int16_t size) {
  return __SHFL_DOWN_SYNC(0xFFFFFFFF, val, delta, size);
}

EXTERN int64_t __kmpc_shuffle_int64(int64_t val, int16_t delta, int16_t size) {
   int lo, hi;
   asm volatile("mov.b64 {%0,%1}, %2;" : "=r"(lo), "=r"(hi) : "l"(val));
   hi = __SHFL_DOWN_SYNC(0xFFFFFFFF, hi, delta, size);
   lo = __SHFL_DOWN_SYNC(0xFFFFFFFF, lo, delta, size);
   asm volatile("mov.b64 %0, {%1,%2};" : "=l"(val) : "r"(lo), "r"(hi));
   return val;
}

INLINE static void gpu_regular_warp_reduce(void *reduce_data,
                                           kmp_ShuffleReductFctPtr shflFct) {
  for (uint32_t mask = WARPSIZE / 2; mask > 0; mask /= 2) {
    shflFct(reduce_data, /*LaneId - not used= */ 0,
            /*Offset = */ mask, /*AlgoVersion=*/0);
  }
}

INLINE static void gpu_irregular_warp_reduce(void *reduce_data,
                                             kmp_ShuffleReductFctPtr shflFct,
                                             uint32_t size, uint32_t tid) {
  uint32_t curr_size;
  uint32_t mask;
  curr_size = size;
  mask = curr_size / 2;
  while (mask > 0) {
    shflFct(reduce_data, /*LaneId = */ tid, /*Offset=*/mask, /*AlgoVersion=*/1);
    curr_size = (curr_size + 1) / 2;
    mask = curr_size / 2;
  }
}

INLINE static uint32_t
gpu_irregular_simd_reduce(void *reduce_data, kmp_ShuffleReductFctPtr shflFct) {
  uint32_t lanemask_lt;
  uint32_t lanemask_gt;
  uint32_t size, remote_id, physical_lane_id;
  physical_lane_id = GetThreadIdInBlock() % WARPSIZE;
  asm("mov.u32 %0, %%lanemask_lt;" : "=r"(lanemask_lt));
  uint32_t Liveness = __ACTIVEMASK();
  uint32_t logical_lane_id = __popc(Liveness & lanemask_lt) * 2;
  asm("mov.u32 %0, %%lanemask_gt;" : "=r"(lanemask_gt));
  do {
    Liveness = __ACTIVEMASK();
    remote_id = __ffs(Liveness & lanemask_gt);
    size = __popc(Liveness);
    logical_lane_id /= 2;
    shflFct(reduce_data, /*LaneId =*/logical_lane_id,
            /*Offset=*/remote_id - 1 - physical_lane_id, /*AlgoVersion=*/2);
  } while (logical_lane_id % 2 == 0 && size > 1);
  return (logical_lane_id == 0);
}

EXTERN
int32_t __kmpc_nvptx_simd_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                        size_t reduce_size, void *reduce_data,
                                        kmp_ShuffleReductFctPtr shflFct,
                                        kmp_InterWarpCopyFctPtr cpyFct) {
  uint32_t Liveness = __ACTIVEMASK();
  if (Liveness == 0xffffffff) {
    gpu_regular_warp_reduce(reduce_data, shflFct);
    return GetThreadIdInBlock() % WARPSIZE ==
           0; // Result on lane 0 of the simd warp.
  } else {
    return gpu_irregular_simd_reduce(
        reduce_data, shflFct); // Result on the first active lane.
  }
}

INLINE
static int32_t nvptx_parallel_reduce_nowait(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    bool isSPMDExecutionMode, bool isRuntimeUninitialized) {
  uint32_t BlockThreadId = GetLogicalThreadIdInBlock(isSPMDExecutionMode);
  uint32_t NumThreads = GetNumberOfOmpThreads(isSPMDExecutionMode);
  if (NumThreads == 1)
    return 1;
  /*
   * This reduce function handles reduction within a team. It handles
   * parallel regions in both L1 and L2 parallelism levels. It also
   * supports Generic, SPMD, and NoOMP modes.
   *
   * 1. Reduce within a warp.
   * 2. Warp master copies value to warp 0 via shared memory.
   * 3. Warp 0 reduces to a single value.
   * 4. The reduced value is available in the thread that returns 1.
   */

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  uint32_t WarpsNeeded = (NumThreads + WARPSIZE - 1) / WARPSIZE;
  uint32_t WarpId = BlockThreadId / WARPSIZE;

  // Volta execution model:
  // For the Generic execution mode a parallel region either has 1 thread and
  // beyond that, always a multiple of 32. For the SPMD execution mode we may
  // have any number of threads.
  if ((NumThreads % WARPSIZE == 0) || (WarpId < WarpsNeeded - 1))
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (NumThreads > 1) // Only SPMD execution mode comes thru this case.
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/NumThreads % WARPSIZE,
                              /*LaneId=*/GetThreadIdInBlock() % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  //
  // Only L1 parallel region can enter this if condition.
  if (NumThreads > WARPSIZE) {
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                BlockThreadId);
  }
  return BlockThreadId == 0;
#else
  uint32_t Liveness = __ACTIVEMASK();
  if (Liveness == 0xffffffff) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (!(Liveness & (Liveness + 1))) // Partial warp but contiguous lanes
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/__popc(Liveness),
                              /*LaneId=*/GetThreadIdInBlock() % WARPSIZE);
  else if (!isRuntimeUninitialized) // Dispersed lanes. Only threads in L2
                                    // parallel region may enter here; return
                                    // early.
    return gpu_irregular_simd_reduce(reduce_data, shflFct);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  //
  // Only L1 parallel region can enter this if condition.
  if (NumThreads > WARPSIZE) {
    uint32_t WarpsNeeded = (NumThreads + WARPSIZE - 1) / WARPSIZE;
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    uint32_t WarpId = BlockThreadId / WARPSIZE;
    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                BlockThreadId);

    return BlockThreadId == 0;
  } else if (isRuntimeUninitialized /* Never an L2 parallel region without the OMP runtime */) {
    return BlockThreadId == 0;
  }

  // Get the OMP thread Id. This is different from BlockThreadId in the case of
  // an L2 parallel region.
  return global_tid == 0;
#endif // __CUDA_ARCH__ >= 700
}

EXTERN __attribute__((deprecated)) int32_t __kmpc_nvptx_parallel_reduce_nowait(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(global_tid, num_vars, reduce_size,
                                      reduce_data, shflFct, cpyFct,
                                      isSPMDMode(), isRuntimeUninitialized());
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_v2(
    kmp_Ident *loc, int32_t global_tid, int32_t num_vars, size_t reduce_size,
    void *reduce_data, kmp_ShuffleReductFctPtr shflFct,
    kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(
      global_tid, num_vars, reduce_size, reduce_data, shflFct, cpyFct,
      checkSPMDMode(loc), checkRuntimeUninitialized(loc));
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_simple_spmd(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(
      global_tid, num_vars, reduce_size, reduce_data, shflFct, cpyFct,
      /*isSPMDExecutionMode=*/true, /*isRuntimeUninitialized=*/true);
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_simple_generic(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(
      global_tid, num_vars, reduce_size, reduce_data, shflFct, cpyFct,
      /*isSPMDExecutionMode=*/false, /*isRuntimeUninitialized=*/true);
}

INLINE
static int32_t nvptx_teams_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                         size_t reduce_size, void *reduce_data,
                                         kmp_ShuffleReductFctPtr shflFct,
                                         kmp_InterWarpCopyFctPtr cpyFct,
                                         kmp_CopyToScratchpadFctPtr scratchFct,
                                         kmp_LoadReduceFctPtr ldFct,
                                         bool isSPMDExecutionMode) {
  uint32_t ThreadId = GetLogicalThreadIdInBlock(isSPMDExecutionMode);
  // In non-generic mode all workers participate in the teams reduction.
  // In generic mode only the team master participates in the teams
  // reduction because the workers are waiting for parallel work.
  uint32_t NumThreads =
      isSPMDExecutionMode ? GetNumberOfOmpThreads(/*isSPMDExecutionMode=*/true)
                          : /*Master thread only*/ 1;
  uint32_t TeamId = GetBlockIdInKernel();
  uint32_t NumTeams = GetNumberOfBlocksInKernel();
  __shared__ volatile bool IsLastTeam;

  // Team masters of all teams write to the scratchpad.
  if (ThreadId == 0) {
    unsigned int *timestamp = GetTeamsReductionTimestamp();
    char *scratchpad = GetTeamsReductionScratchpad();

    scratchFct(reduce_data, scratchpad, TeamId, NumTeams);
    __threadfence();

    // atomicInc increments 'timestamp' and has a range [0, NumTeams-1].
    // It resets 'timestamp' back to 0 once the last team increments
    // this counter.
    unsigned val = atomicInc(timestamp, NumTeams - 1);
    IsLastTeam = val == NumTeams - 1;
  }

  // We have to wait on L1 barrier because in GENERIC mode the workers
  // are waiting on barrier 0 for work.
  //
  // If we guard this barrier as follows it leads to deadlock, probably
  // because of a compiler bug: if (!IsGenericMode()) __syncthreads();
  uint16_t SyncWarps = (NumThreads + WARPSIZE - 1) / WARPSIZE;
  named_sync(L1_BARRIER, SyncWarps * WARPSIZE);

  // If this team is not the last, quit.
  if (/* Volatile read by all threads */ !IsLastTeam)
    return 0;

    //
    // Last team processing.
    //

    // Threads in excess of #teams do not participate in reduction of the
    // scratchpad values.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  uint32_t ActiveThreads = NumThreads;
  if (NumTeams < NumThreads) {
    ActiveThreads =
        (NumTeams < WARPSIZE) ? 1 : NumTeams & ~((uint16_t)WARPSIZE - 1);
  }
  if (ThreadId >= ActiveThreads)
    return 0;

  // Load from scratchpad and reduce.
  char *scratchpad = GetTeamsReductionScratchpad();
  ldFct(reduce_data, scratchpad, ThreadId, NumTeams, /*Load only*/ 0);
  for (uint32_t i = ActiveThreads + ThreadId; i < NumTeams; i += ActiveThreads)
    ldFct(reduce_data, scratchpad, i, NumTeams, /*Load and reduce*/ 1);

  uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
  uint32_t WarpId = ThreadId / WARPSIZE;

  // Reduce across warps to the warp master.
  if ((ActiveThreads % WARPSIZE == 0) ||
      (WarpId < WarpsNeeded - 1)) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (ActiveThreads > 1) // Partial warp but contiguous lanes
    // Only SPMD execution mode comes thru this case.
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/ActiveThreads % WARPSIZE,
                              /*LaneId=*/ThreadId % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  if (ActiveThreads > WARPSIZE) {
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded, ThreadId);
  }
#else
  if (ThreadId >= NumTeams)
    return 0;

  // Load from scratchpad and reduce.
  char *scratchpad = GetTeamsReductionScratchpad();
  ldFct(reduce_data, scratchpad, ThreadId, NumTeams, /*Load only*/ 0);
  for (uint32_t i = NumThreads + ThreadId; i < NumTeams; i += NumThreads)
    ldFct(reduce_data, scratchpad, i, NumTeams, /*Load and reduce*/ 1);

  // Reduce across warps to the warp master.
  uint32_t Liveness = __ACTIVEMASK();
  if (Liveness == 0xffffffff) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else // Partial warp but contiguous lanes
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/__popc(Liveness),
                              /*LaneId=*/ThreadId % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  uint32_t ActiveThreads = NumTeams < NumThreads ? NumTeams : NumThreads;
  if (ActiveThreads > WARPSIZE) {
    uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    uint32_t WarpId = ThreadId / WARPSIZE;
    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded, ThreadId);
  }
#endif // __CUDA_ARCH__ >= 700

  return ThreadId == 0;
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                         size_t reduce_size, void *reduce_data,
                                         kmp_ShuffleReductFctPtr shflFct,
                                         kmp_InterWarpCopyFctPtr cpyFct,
                                         kmp_CopyToScratchpadFctPtr scratchFct,
                                         kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, isSPMDMode());
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait_simple_spmd(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, /*isSPMDExecutionMode=*/true);
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait_simple_generic(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, /*isSPMDExecutionMode=*/false);
}

EXTERN int32_t __kmpc_nvptx_teams_reduce_nowait_simple(kmp_Ident *loc,
                                                       int32_t global_tid,
                                                       kmp_CriticalName *crit) {
  if (checkSPMDMode(loc) && GetThreadIdInBlock() != 0)
    return 0;
  // The master thread of the team actually does the reduction.
  while (atomicCAS((uint32_t *)crit, 0, 1))
    ;
  return 1;
}

EXTERN void
__kmpc_nvptx_teams_end_reduce_nowait_simple(kmp_Ident *loc, int32_t global_tid,
                                            kmp_CriticalName *crit) {
  __threadfence_system();
  (void)atomicExch((uint32_t *)crit, 0);
}

INLINE static bool isMaster(kmp_Ident *loc, uint32_t ThreadId) {
  return checkGenericMode(loc) || IsTeamMaster(ThreadId);
}

INLINE static uint32_t roundToWarpsize(uint32_t s) {
  if (s < WARPSIZE)
    return 1;
  return (s & ~(unsigned)(WARPSIZE - 1));
}

__device__ static volatile uint32_t IterCnt = 0;
__device__ static volatile uint32_t Cnt = 0;
EXTERN int32_t __kmpc_nvptx_teams_reduce_nowait_v2(
    kmp_Ident *loc, int32_t global_tid, void *global_buffer,
    int32_t num_of_records, void *reduce_data, kmp_ShuffleReductFctPtr shflFct,
    kmp_InterWarpCopyFctPtr cpyFct, kmp_ListGlobalFctPtr lgcpyFct,
    kmp_ListGlobalFctPtr lgredFct, kmp_ListGlobalFctPtr glcpyFct,
    kmp_ListGlobalFctPtr glredFct) {

  // Terminate all threads in non-SPMD mode except for the master thread.
  if (checkGenericMode(loc) && GetThreadIdInBlock() != GetMasterThreadID())
    return 0;

  uint32_t ThreadId = GetLogicalThreadIdInBlock(checkSPMDMode(loc));

  // In non-generic mode all workers participate in the teams reduction.
  // In generic mode only the team master participates in the teams
  // reduction because the workers are waiting for parallel work.
  uint32_t NumThreads =
      checkSPMDMode(loc) ? GetNumberOfOmpThreads(/*isSPMDExecutionMode=*/true)
                         : /*Master thread only*/ 1;
  uint32_t TeamId = GetBlockIdInKernel();
  uint32_t NumTeams = GetNumberOfBlocksInKernel();
  __shared__ unsigned Bound;
  __shared__ unsigned ChunkTeamCount;

  // Block progress for teams greater than the current upper
  // limit. We always only allow a number of teams less or equal
  // to the number of slots in the buffer.
  bool IsMaster = isMaster(loc, ThreadId);
  while (IsMaster) {
    // Atomic read
    Bound = atomicAdd((uint32_t *)&IterCnt, 0);
    if (TeamId < Bound + num_of_records)
      break;
  }

  if (IsMaster) {
    int ModBockId = TeamId % num_of_records;
    if (TeamId < num_of_records)
      lgcpyFct(global_buffer, ModBockId, reduce_data);
    else
      lgredFct(global_buffer, ModBockId, reduce_data);
    __threadfence_system();

    // Increment team counter.
    // This counter is incremented by all teams in the current
    // BUFFER_SIZE chunk.
    ChunkTeamCount = atomicInc((uint32_t *)&Cnt, num_of_records - 1);
  }
  // Synchronize
  if (checkSPMDMode(loc))
    __kmpc_barrier(loc, global_tid);

  // reduce_data is global or shared so before being reduced within the
  // warp we need to bring it in local memory:
  // local_reduce_data = reduce_data[i]
  //
  // Example for 3 reduction variables a, b, c (of potentially different
  // types):
  //
  // buffer layout (struct of arrays):
  // a, a, ..., a, b, b, ... b, c, c, ... c
  // |__________|
  //     num_of_records
  //
  // local_data_reduce layout (struct):
  // a, b, c
  //
  // Each thread will have a local struct containing the values to be
  // reduced:
  //      1. do reduction within each warp.
  //      2. do reduction across warps.
  //      3. write the final result to the main reduction variable
  //         by returning 1 in the thread holding the reduction result.

  // Check if this is the very last team.
  unsigned NumRecs = min(NumTeams, num_of_records);
  if (ChunkTeamCount == NumTeams - Bound - 1) {
    //
    // Last team processing.
    //
    if (ThreadId >= NumRecs)
      return 0;
    NumThreads = roundToWarpsize(min(NumThreads, NumRecs));
    if (ThreadId >= NumThreads)
      return 0;

    // Load from buffer and reduce.
    glcpyFct(global_buffer, ThreadId, reduce_data);
    for (uint32_t i = NumThreads + ThreadId; i < NumRecs; i += NumThreads)
      glredFct(global_buffer, i, reduce_data);

    // Reduce across warps to the warp master.
    if (NumThreads > 1) {
      gpu_regular_warp_reduce(reduce_data, shflFct);

      // When we have more than [warpsize] number of threads
      // a block reduction is performed here.
      uint32_t ActiveThreads = min(NumRecs, NumThreads);
      if (ActiveThreads > WARPSIZE) {
        uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
        // Gather all the reduced values from each warp
        // to the first warp.
        cpyFct(reduce_data, WarpsNeeded);

        uint32_t WarpId = ThreadId / WARPSIZE;
        if (WarpId == 0)
          gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                    ThreadId);
      }
    }

    if (IsMaster) {
      Cnt = 0;
      IterCnt = 0;
      return 1;
    }
    return 0;
  }
  if (IsMaster && ChunkTeamCount == num_of_records - 1) {
    // Allow SIZE number of teams to proceed writing their
    // intermediate results to the global buffer.
    atomicAdd((uint32_t *)&IterCnt, num_of_records);
  }

  return 0;
}

