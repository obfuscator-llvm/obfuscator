#include "hip/hip_runtime.h"
//===------------ omp_data.cu - NVPTX OpenMP GPU objects --------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the data objects used on the GPU device.
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// global device envrionment
////////////////////////////////////////////////////////////////////////////////

__device__ omptarget_device_environmentTy omptarget_device_environment;

////////////////////////////////////////////////////////////////////////////////
// global data holding OpenMP state information
////////////////////////////////////////////////////////////////////////////////

__device__
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

__device__ omptarget_nvptx_SimpleMemoryManager
    omptarget_nvptx_simpleMemoryManager;
__device__ __shared__ uint32_t usedMemIdx;
__device__ __shared__ uint32_t usedSlotIdx;

__device__ __shared__ uint8_t parallelLevel[MAX_THREADS_PER_TEAM / WARPSIZE];
__device__ __shared__ uint16_t threadLimit;
__device__ __shared__ uint16_t threadsInTeam;
__device__ __shared__ uint16_t nThreads;
// Pointer to this team's OpenMP state object
__device__ __shared__
    omptarget_nvptx_ThreadPrivateContext *omptarget_nvptx_threadPrivateContext;

////////////////////////////////////////////////////////////////////////////////
// The team master sets the outlined parallel function in this variable to
// communicate with the workers.  Since it is in shared memory, there is one
// copy of these variables for each kernel, instance, and team.
////////////////////////////////////////////////////////////////////////////////
volatile __device__ __shared__ omptarget_nvptx_WorkFn omptarget_nvptx_workFn;

////////////////////////////////////////////////////////////////////////////////
// OpenMP kernel execution parameters
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ uint32_t execution_param;

////////////////////////////////////////////////////////////////////////////////
// Data sharing state
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ DataSharingStateTy DataSharingState;

////////////////////////////////////////////////////////////////////////////////
// Scratchpad for teams reduction.
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ void *ReductionScratchpadPtr;

////////////////////////////////////////////////////////////////////////////////
// Data sharing related variables.
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ omptarget_nvptx_SharedArgs omptarget_nvptx_globalArgs;
