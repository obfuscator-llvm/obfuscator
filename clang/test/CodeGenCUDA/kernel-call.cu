#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -target-sdk-version=8.0 -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=CUDA-OLD,CHECK
// RUN: %clang_cc1 -target-sdk-version=9.2  -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=CUDA-NEW,CHECK
// RUN: %clang_cc1 -x hip -emit-llvm %s -o - \
// RUN: | FileCheck %s --check-prefixes=HIP,CHECK


#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: define{{.*}}g1
// HIP: call{{.*}}hipSetupArgument
// HIP: call{{.*}}hipLaunchByPtr
// CUDA-OLD: call{{.*}}hipSetupArgument
// CUDA-OLD: call{{.*}}hipLaunchByPtr
// CUDA-NEW: call{{.*}}__cudaPopCallConfiguration
// CUDA-NEW: call{{.*}}hipLaunchKernel
__global__ void g1(int x) {}

// CHECK-LABEL: define{{.*}}main
int main(void) {
  // HIP: call{{.*}}hipConfigureCall
  // CUDA-OLD: call{{.*}}hipConfigureCall
  // CUDA-NEW: call{{.*}}__cudaPushCallConfiguration
  // CHECK: icmp
  // CHECK: br
  // CHECK: call{{.*}}g1
  g1<<<1, 1>>>(42);
}
