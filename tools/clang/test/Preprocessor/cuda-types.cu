
#include <hip/hip_runtime.h>
// Check that types, widths, __GCC_ATOMIC* macros, etc. match on the host and
// device sides of CUDA compilations.  Note that we filter out long double, as
// this is intentionally different on host and device.
//
// FIXME: We really should make __GCC_HAVE_SYNC_COMPARE_AND_SWAP identical on
// host and device, but architecturally this is difficult at the moment.

// RUN: %clang --cuda-host-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-device-defines-filtered
// RUN: diff %T/i386-host-defines-filtered %T/i386-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-device-defines-filtered
// RUN: diff %T/x86_64-host-defines-filtered %T/x86_64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/powerpc64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/powerpc64-device-defines-filtered
// RUN: diff %T/powerpc64-host-defines-filtered %T/powerpc64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target i386-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-msvc-device-defines-filtered
// RUN: diff %T/i386-msvc-host-defines-filtered %T/i386-msvc-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)\|define __GCC_ATOMIC' \
// RUN:   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-msvc-device-defines-filtered
// RUN: diff %T/x86_64-msvc-host-defines-filtered %T/x86_64-msvc-device-defines-filtered
