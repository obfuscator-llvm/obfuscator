#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

__device__ void foo() {
  extern __shared__ int x; // expected-error {{__shared__ variable 'x' cannot be 'extern'}}
  extern __shared__ int arr[];  // ok
  extern __shared__ int arr0[0]; // expected-error {{__shared__ variable 'arr0' cannot be 'extern'}}
  extern __shared__ int arr1[1]; // expected-error {{__shared__ variable 'arr1' cannot be 'extern'}}
  extern __shared__ int* ptr ; // expected-error {{__shared__ variable 'ptr' cannot be 'extern'}}
}

__host__ __device__ void bar() {
  extern __shared__ int arr[];  // ok
  extern __shared__ int arr0[0]; // expected-error {{__shared__ variable 'arr0' cannot be 'extern'}}
  extern __shared__ int arr1[1]; // expected-error {{__shared__ variable 'arr1' cannot be 'extern'}}
  extern __shared__ int* ptr ; // expected-error {{__shared__ variable 'ptr' cannot be 'extern'}}
}

extern __shared__ int global; // expected-error {{__shared__ variable 'global' cannot be 'extern'}}
extern __shared__ int global_arr[]; // ok
extern __shared__ int global_arr1[1]; // expected-error {{__shared__ variable 'global_arr1' cannot be 'extern'}}
