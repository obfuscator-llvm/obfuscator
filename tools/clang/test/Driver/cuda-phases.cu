
#include <hip/hip_runtime.h>
// Tests the phases generated for a CUDA offloading target for different
// combinations of:
// - Number of gpu architectures;
// - Host/device-only compilation;
// - User-requested final phase - binary or assembly.

// REQUIRES: clang-driver
// REQUIRES: powerpc-registered-target
// REQUIRES: nvptx-registered-target

//
// Test single gpu architecture with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefix=BIN %s
// BIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// BIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (host-cuda)
// BIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-cuda)
// BIN-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// BIN-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, cuda-cpp-output, (device-cuda, sm_30)
// BIN-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-cuda, sm_30)
// BIN-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-cuda, sm_30)
// BIN-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-cuda, sm_30)
// BIN-DAG: [[P8:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P7]]}, object
// BIN-DAG: [[P9:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P6]]}, assembler
// BIN-DAG: [[P10:[0-9]+]]: linker, {[[P8]], [[P9]]}, cuda-fatbin, (device-cuda)
// BIN-DAG: [[P11:[0-9]+]]: offload, "host-cuda (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-cuda (nvptx64-nvidia-cuda)" {[[P10]]}, ir
// BIN-DAG: [[P12:[0-9]+]]: backend, {[[P11]]}, assembler, (host-cuda)
// BIN-DAG: [[P13:[0-9]+]]: assembler, {[[P12]]}, object, (host-cuda)
// BIN-DAG: [[P14:[0-9]+]]: linker, {[[P13]]}, image, (host-cuda)

//
// Test single gpu architecture up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 %s -S 2>&1 \
// RUN: | FileCheck -check-prefix=ASM %s
// ASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// ASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (device-cuda, sm_30)
// ASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-cuda, sm_30)
// ASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-cuda, sm_30)
// ASM-DAG: [[P4:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P3]]}, assembler
// ASM-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// ASM-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, cuda-cpp-output, (host-cuda)
// ASM-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (host-cuda)
// ASM-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (host-cuda)

//
// Test two gpu architectures with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefix=BIN2 %s
// BIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// BIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (host-cuda)
// BIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-cuda)
// BIN2-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// BIN2-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, cuda-cpp-output, (device-cuda, sm_30)
// BIN2-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-cuda, sm_30)
// BIN2-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-cuda, sm_30)
// BIN2-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-cuda, sm_30)
// BIN2-DAG: [[P8:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P7]]}, object
// BIN2-DAG: [[P9:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P6]]}, assembler
// BIN2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_35)
// BIN2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, cuda-cpp-output, (device-cuda, sm_35)
// BIN2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (device-cuda, sm_35)
// BIN2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (device-cuda, sm_35)
// BIN2-DAG: [[P14:[0-9]+]]: assembler, {[[P13]]}, object, (device-cuda, sm_35)
// BIN2-DAG: [[P15:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_35)" {[[P14]]}, object
// BIN2-DAG: [[P16:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_35)" {[[P13]]}, assembler
// BIN2-DAG: [[P17:[0-9]+]]: linker, {[[P8]], [[P9]], [[P15]], [[P16]]}, cuda-fatbin, (device-cuda)
// BIN2-DAG: [[P18:[0-9]+]]: offload, "host-cuda (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-cuda (nvptx64-nvidia-cuda)" {[[P17]]}, ir
// BIN2-DAG: [[P19:[0-9]+]]: backend, {[[P18]]}, assembler, (host-cuda)
// BIN2-DAG: [[P20:[0-9]+]]: assembler, {[[P19]]}, object, (host-cuda)
// BIN2-DAG: [[P21:[0-9]+]]: linker, {[[P20]]}, image, (host-cuda)

//
// Test two gpu architecturess up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -S 2>&1 \
// RUN: | FileCheck -check-prefix=ASM2 %s
// ASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// ASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (device-cuda, sm_30)
// ASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-cuda, sm_30)
// ASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-cuda, sm_30)
// ASM2-DAG: [[P4:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P3]]}, assembler
// ASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_35)
// ASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, cuda-cpp-output, (device-cuda, sm_35)
// ASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-cuda, sm_35)
// ASM2-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-cuda, sm_35)
// ASM2-DAG: [[P9:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_35)" {[[P8]]}, assembler
// ASM2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// ASM2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, cuda-cpp-output, (host-cuda)
// ASM2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (host-cuda)
// ASM2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (host-cuda)

//
// Test single gpu architecture with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefix=HBIN %s
// HBIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// HBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (host-cuda)
// HBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-cuda)
// HBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-cuda)
// HBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-cuda)
// HBIN-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-cuda)
//
// Test single gpu architecture up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=HASM %s
// HASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// HASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (host-cuda)
// HASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-cuda)
// HASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-cuda)

//
// Test two gpu architectures with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefix=HBIN2 %s
// HBIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// HBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (host-cuda)
// HBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-cuda)
// HBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-cuda)
// HBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-cuda)
// HBIN2-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-cuda)

//
// Test two gpu architectures up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=HASM2 %s
// HASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (host-cuda)
// HASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (host-cuda)
// HASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-cuda)
// HASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-cuda)

//
// Test single gpu architecture with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefix=DBIN %s
// DBIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// DBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (device-cuda, sm_30)
// DBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-cuda, sm_30)
// DBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-cuda, sm_30)
// DBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-cuda, sm_30)
// DBIN-DAG: [[P5:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P4]]}, object

//
// Test single gpu architecture up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=DASM %s
// DASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// DASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (device-cuda, sm_30)
// DASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-cuda, sm_30)
// DASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-cuda, sm_30)
// DASM-DAG: [[P4:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P3]]}, assembler

//
// Test two gpu architectures with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefix=DBIN2 %s
// DBIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// DBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (device-cuda, sm_30)
// DBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-cuda, sm_30)
// DBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-cuda, sm_30)
// DBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-cuda, sm_30)
// DBIN2-DAG: [[P5:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P4]]}, object
// DBIN2-DAG: [[P6:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_35)
// DBIN2-DAG: [[P7:[0-9]+]]: preprocessor, {[[P6]]}, cuda-cpp-output, (device-cuda, sm_35)
// DBIN2-DAG: [[P8:[0-9]+]]: compiler, {[[P7]]}, ir, (device-cuda, sm_35)
// DBIN2-DAG: [[P9:[0-9]+]]: backend, {[[P8]]}, assembler, (device-cuda, sm_35)
// DBIN2-DAG: [[P10:[0-9]+]]: assembler, {[[P9]]}, object, (device-cuda, sm_35)
// DBIN2-DAG: [[P11:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_35)" {[[P10]]}, object

//
// Test two gpu architectures up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=DASM2 %s
// DASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_30)
// DASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, cuda-cpp-output, (device-cuda, sm_30)
// DASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-cuda, sm_30)
// DASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-cuda, sm_30)
// DASM2-DAG: [[P4:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_30)" {[[P3]]}, assembler
// DASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", cuda, (device-cuda, sm_35)
// DASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, cuda-cpp-output, (device-cuda, sm_35)
// DASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-cuda, sm_35)
// DASM2-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-cuda, sm_35)
// DASM2-DAG: [[P9:[0-9]+]]: offload, "device-cuda (nvptx64-nvidia-cuda:sm_35)" {[[P8]]}, assembler
