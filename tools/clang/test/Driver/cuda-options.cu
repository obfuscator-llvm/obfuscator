
#include <hip/hip_runtime.h>
// Tests CUDA compilation pipeline construction in Driver.
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Simple compilation case. Compile device-side to PTX assembly and make sure
// we use it on the host side.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Typical compilation + link case.
// RUN: %clang -### -target x86_64-linux-gnu %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-host-only disables device-side compilation, but doesn't
// disable host-side compilation/linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// Verify that --cuda-device-only disables host-side compilation and linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// Check that the last of --cuda-compile-host-device, --cuda-host-only, and
// --cuda-device-only wins.

// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:    --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only \
// RUN:    --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only \
// RUN:   --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-gpu-arch option passes the correct GPU archtecture to
// device compilation.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE-SM30 -check-prefix HOST \
// RUN:    -check-prefix INCLUDES-DEVICE -check-prefix NOLINK %s

// Verify that there is one device-side compilation per --cuda-gpu-arch args
// and that all results are included on the host side.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE2 -check-prefix DEVICE-SM30 \
// RUN:    -check-prefix DEVICE2-SM35 -check-prefix HOST \
// RUN:    -check-prefix HOST-NOSAVE -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -save-temps is used.
// RUN: %clang -### -target x86_64-linux-gnu -save-temps -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-SAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-SAVE -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -fno-integrated-as is used.
// RUN: %clang -### -target x86_64-linux-gnu -fno-integrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-NOSAVE \
// RUN:    -check-prefix HOST-AS -check-prefix NOLINK %s

// Verify that --[no-]cuda-gpu-arch arguments are handled correctly.
// a) --no-cuda-gpu-arch=X negates preceeding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=sm_35 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM20,ARCH-SM30,NOARCH-SM35 %s

// b) --no-cuda-gpu-arch=X negates more than one preceeding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=sm_35 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM20,ARCH-SM30,NOARCH-SM35 %s

// c) if --no-cuda-gpu-arch=X negates all preceeding --cuda-gpu-arch=X
//    we default to sm_20 -- same as if no --cuda-gpu-arch were passed.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=sm_35 --no-cuda-gpu-arch=sm_30 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM20,NOARCH-SM30,NOARCH-SM35 %s

// d) --no-cuda-gpu-arch=X is a no-op if there's no preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30\
// RUN:   --no-cuda-gpu-arch=sm_50 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM20,ARCH-SM30,ARCH-SM35 %s

// e) --no-cuda-gpu-arch=X does not affect following --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --no-cuda-gpu-arch=sm_35 --no-cuda-gpu-arch=sm_30 \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM20,ARCH-SM30,ARCH-SM35 %s

// f) --no-cuda-gpu-arch=all negates all preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_20 --cuda-gpu-arch=sm_30 \
// RUN:   --no-cuda-gpu-arch=all \
// RUN:   --cuda-gpu-arch=sm_35 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM20,NOARCH-SM30,ARCH-SM35 %s

// g) There's no --cuda-gpu-arch=all
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCHALLERROR %s

// ARCH-SM20: "-cc1"{{.*}}"-target-cpu" "sm_20"
// NOARCH-SM20-NOT: "-cc1"{{.*}}"-target-cpu" "sm_20"
// ARCH-SM30: "-cc1"{{.*}}"-target-cpu" "sm_30"
// NOARCH-SM30-NOT: "-cc1"{{.*}}"-target-cpu" "sm_30"
// ARCH-SM35: "-cc1"{{.*}}"-target-cpu" "sm_35"
// NOARCH-SM35-NOT: "-cc1"{{.*}}"-target-cpu" "sm_35"
// ARCHALLERROR: error: Unsupported CUDA gpu architecture: all

// Match device-side preprocessor and compiler phases with -save-temps.
// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda"

// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda-cpp-output"

// Match the job that produces PTX assembly.
// DEVICE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-NOSAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAME: "-fcuda-is-device"
// DEVICE-SM30-SAME: "-target-cpu" "sm_30"
// DEVICE-SAME: "-o" "[[PTXFILE:[^"]*]]"
// DEVICE-NOSAVE-SAME: "-x" "cuda"
// DEVICE-SAVE-SAME: "-x" "ir"

// Match the call to ptxas (which assembles PTX to SASS).
// DEVICE:ptxas
// DEVICE-SM30-DAG: "--gpu-name" "sm_30"
// DEVICE-DAG: "--output-file" "[[CUBINFILE:[^"]*]]"
// DEVICE-DAG: "[[PTXFILE]]"

// Match another device-side compilation.
// DEVICE2: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE2-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE2-SAME: "-fcuda-is-device"
// DEVICE2-SM35-SAME: "-target-cpu" "sm_35"
// DEVICE2-SAME: "-o" "[[GPUBINARY2:[^"]*]]"
// DEVICE2-SAME: "-x" "cuda"

// Match no device-side compilation.
// NODEVICE-NOT: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// NODEVICE-NOT: "-fcuda-is-device"

// INCLUDES-DEVICE:fatbinary
// INCLUDES-DEVICE-DAG: "--create" "[[FATBINARY:[^"]*]]"
// INCLUDES-DEVICE-DAG: "--image=profile=sm_{{[0-9]+}},file=[[CUBINFILE]]"
// INCLUDES-DEVICE-DAG: "--image=profile=compute_{{[0-9]+}},file=[[PTXFILE]]"

// Match host-side preprocessor job with -save-temps.
// HOST-SAVE: "-cc1" "-triple" "x86_64--linux-gnu"
// HOST-SAVE-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-SAVE-NOT: "-fcuda-is-device"
// HOST-SAVE-SAME: "-x" "cuda"

// Match host-side compilation.
// HOST: "-cc1" "-triple" "x86_64--linux-gnu"
// HOST-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-NOT: "-fcuda-is-device"
// HOST-SAME: "-o" "[[HOSTOUTPUT:[^"]*]]"
// HOST-NOSAVE-SAME: "-x" "cuda"
// HOST-SAVE-SAME: "-x" "cuda-cpp-output"
// INCLUDES-DEVICE-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"

// Match external assembler that uses compilation output.
// HOST-AS: "-o" "{{.*}}.o" "[[HOSTOUTPUT]]"

// Match no GPU code inclusion.
// NOINCLUDES-DEVICE-NOT: "-fcuda-include-gpubinary"

// Match no host compilation.
// NOHOST-NOT: "-cc1" "-triple"
// NOHOST-NOT: "-x" "cuda"

// Match linker.
// LINK: "{{.*}}{{ld|link}}{{(.exe)?}}"
// LINK-SAME: "[[HOSTOUTPUT]]"

// Match no linker.
// NOLINK-NOT: "{{.*}}{{ld|link}}{{(.exe)?}}"
