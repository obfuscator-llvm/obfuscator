
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// # Check that we properly detect CUDA installation.
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/no-cuda-there 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --sysroot=%S/no-cuda-there 2>&1 | FileCheck %s -check-prefix NOCUDA

// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/Inputs/CUDA 2>&1 | FileCheck %s
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --sysroot=%S/Inputs/CUDA 2>&1 | FileCheck %s

// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s
// RUN: %clang -v --target=i386-apple-macosx \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s

// Make sure we map libdevice bitcode files to proper GPUs. These
// tests use Inputs/CUDA_80 which has full set of libdevice files.
// However, libdevice mapping only matches CUDA-7.x at the moment.
// sm_2x, sm_32 -> compute_20
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_21 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE20
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_32 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE20
// sm_30, sm_6x map to compute_30.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_30 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE30
// sm_5x is a special case. Maps to compute_30 for cuda-7.x only.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE30
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_60 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE30
// sm_35 and sm_37 -> compute_35
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix CUDAINC \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE35
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_37 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix CUDAINC \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE35
// sm_5x -> compute_50 for CUDA-8.0 and newer.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_50 \
// RUN:   --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE50

// Verify that -nocudainc prevents adding include path to CUDA headers.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE35
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE35

// We should not add any CUDA include paths if there's no valid CUDA installation
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC

// Verify that we get an error if there's no libdevice library to link with.
// NOTE: Inputs/CUDA deliberately does *not* have libdevice.compute_20  for this purpose.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix MISSINGLIBDEVICE
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix MISSINGLIBDEVICE

// Verify that  -nocudalib prevents linking libdevice bitcode in.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudalib --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOLIBDEVICE
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   -nocudalib --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOLIBDEVICE

// Verify that we don't add include paths, link with libdevice or
// -include __clang_cuda_runtime_wrapper.h without valid CUDA installation.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix NOCUDAINC -check-prefix NOLIBDEVICE
// RUN: %clang -### -v --target=i386-apple-macosx --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix NOCUDAINC -check-prefix NOLIBDEVICE

// Verify that C++ include paths are passed for both host and device frontends.
// RUN: %clang -### -no-canonical-prefixes -target x86_64-linux-gnu %s \
// RUN: --stdlib=libstdc++ --sysroot=%S/Inputs/ubuntu_14.04_multiarch_tree2 \
// RUN: --gcc-toolchain="" 2>&1 \
// RUN: | FileCheck %s --check-prefix CHECK-CXXINCLUDE

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// NOCUDA-NOT: Found CUDA installation:

// MISSINGLIBDEVICE: error: cannot find libdevice for sm_20.

// COMMON: "-triple" "nvptx-nvidia-cuda"
// COMMON-SAME: "-fcuda-is-device"
// LIBDEVICE-SAME: "-mlink-cuda-bitcode"
// NOLIBDEVICE-NOT: "-mlink-cuda-bitcode"
// LIBDEVICE20-SAME: libdevice.compute_20.10.bc
// LIBDEVICE30-SAME: libdevice.compute_30.10.bc
// LIBDEVICE35-SAME: libdevice.compute_35.10.bc
// LIBDEVICE50-SAME: libdevice.compute_50.10.bc
// NOLIBDEVICE-NOT: libdevice.compute_{{.*}}.bc
// LIBDEVICE-SAME: "-target-feature" "+ptx42"
// NOLIBDEVICE-NOT: "-target-feature" "+ptx42"
// CUDAINC-SAME: "-internal-isystem" "{{.*}}/Inputs/CUDA{{[_0-9]+}}/usr/local/cuda/include"
// NOCUDAINC-NOT: "-internal-isystem" "{{.*}}/cuda/include"
// CUDAINC-SAME: "-include" "__clang_cuda_runtime_wrapper.h"
// NOCUDAINC-NOT: "-include" "__clang_cuda_runtime_wrapper.h"
// -internal-externc-isystem flags must come *after* the cuda include flags,
// because we must search the cuda include directory first.
// CUDAINC-SAME: "-internal-externc-isystem"
// COMMON-SAME: "-x" "cuda"
// CHECK-CXXINCLUDE: clang{{.*}} "-cc1" "-triple" "nvptx64-nvidia-cuda"
// CHECK-CXXINCLUDE-SAME: {{.*}}"-internal-isystem" "{{.+}}/include/c++/4.8"
// CHECK-CXXINCLUDE: clang{{.*}} "-cc1" "-triple" "x86_64--linux-gnu"
// CHECK-CXXINCLUDE-SAME: {{.*}}"-internal-isystem" "{{.+}}/include/c++/4.8"
// CHECK-CXXINCLUDE: ld{{.*}}"
