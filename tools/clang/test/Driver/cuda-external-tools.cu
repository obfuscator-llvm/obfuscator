
#include <hip/hip_runtime.h>
// Tests that ptxas and fatbinary are invoked correctly during CUDA
// compilation.
//
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Regular compiles with -O{0,1,2,3,4,fast}.  -O4 and -Ofast map to ptxas O3.
// RUN: %clang -### -target x86_64-linux-gnu -O0 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT0 %s
// RUN: %clang -### -target x86_64-linux-gnu -O1 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT1 %s
// RUN: %clang -### -target x86_64-linux-gnu -O2 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT2 %s
// RUN: %clang -### -target x86_64-linux-gnu -O3 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT3 %s
// RUN: %clang -### -target x86_64-linux-gnu -O4 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT3 %s
// RUN: %clang -### -target x86_64-linux-gnu -Ofast -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT3 %s

// With debugging enabled, ptxas should be run with with no ptxas optimizations.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-noopt-device-debug -O2 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix DBG %s

// --no-cuda-noopt-device-debug overrides --cuda-noopt-device-debug.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-noopt-device-debug \
// RUN:   --no-cuda-noopt-device-debug -O2 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT2 %s

// Regular compile without -O.  This should result in us passing -O0 to ptxas.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT0 %s

// Regular compiles with -Os and -Oz.  For lack of a better option, we map
// these to ptxas -O3.
// RUN: %clang -### -target x86_64-linux-gnu -Os -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT2 %s
// RUN: %clang -### -target x86_64-linux-gnu -Oz -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT2 %s

// Regular compile targeting sm_35.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM35 %s

// 32-bit compile.
// RUN: %clang -### -target x86_32-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH32 -check-prefix SM20 %s

// Compile with -fintegrated-as.  This should still cause us to invoke ptxas.
// RUN: %clang -### -target x86_64-linux-gnu -fintegrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT0 %s

// Check -Xcuda-ptxas and -Xcuda-fatbinary
// RUN: %clang -### -target x86_64-linux-gnu -c -Xcuda-ptxas -foo1 \
// RUN:   -Xcuda-fatbinary -bar1 -Xcuda-ptxas -foo2 -Xcuda-fatbinary -bar2 %s 2>&1 \
// RUN: | FileCheck -check-prefix SM20 -check-prefix PTXAS-EXTRA \
// RUN:   -check-prefix FATBINARY-EXTRA %s

// MacOS spot-checks
// RUN: %clang -### -target x86_64-apple-macosx -O0 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 -check-prefix OPT0 %s
// RUN: %clang -### -target x86_64-apple-macosx --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM35 %s
// RUN: %clang -### -target x86_32-apple-macosx -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH32 -check-prefix SM20 %s

// Match clang job that produces PTX assembly.
// CHECK: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// SM20: "-target-cpu" "sm_20"
// SM35: "-target-cpu" "sm_35"
// SM20: "-o" "[[PTXFILE:[^"]*]]"
// SM35: "-o" "[[PTXFILE:[^"]*]]"

// Match the call to ptxas (which assembles PTX to SASS).
// CHECK:ptxas
// ARCH64: "-m64"
// ARCH32: "-m32"
// OPT0: "-O0"
// OPT0-NOT: "-g"
// OPT1: "-O1"
// OPT1-NOT: "-g"
// OPT2: "-O2"
// OPT2-NOT: "-g"
// OPT3: "-O3"
// OPT3-NOT: "-g"
// DBG: "-g" "--dont-merge-basicblocks" "--return-at-end"
// SM20: "--gpu-name" "sm_20"
// SM35: "--gpu-name" "sm_35"
// SM20: "--output-file" "[[CUBINFILE:[^"]*]]"
// SM35: "--output-file" "[[CUBINFILE:[^"]*]]"
// PTXAS-EXTRA: "-foo1"
// PTXAS-EXTRA-SAME: "-foo2"
// CHECK-SAME: "[[PTXFILE]]"

// Match the call to fatbinary (which combines all our PTX and SASS into one
// blob).
// CHECK:fatbinary
// CHECK-DAG: "--cuda"
// ARCH64-DAG: "-64"
// ARCH32-DAG: "-32"
// CHECK-DAG: "--create" "[[FATBINARY:[^"]*]]"
// SM20-DAG: "--image=profile=compute_20,file=[[PTXFILE]]"
// SM35-DAG: "--image=profile=compute_35,file=[[PTXFILE]]"
// SM20-DAG: "--image=profile=sm_20,file=[[CUBINFILE]]"
// SM35-DAG: "--image=profile=sm_35,file=[[CUBINFILE]]"
// FATBINARY-EXTRA: "-bar1"
// FATBINARY-EXTRA-SAME: "-bar2"

// Match the clang job for host compilation.
// CHECK: "-cc1" "-triple" "x86_64--linux-gnu"
// CHECK-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"
