
#include <hip/hip_runtime.h>
// Tests that "sm_XX" gets correctly converted to "compute_YY" when we invoke
// fatbinary.
//
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// CHECK:fatbinary

// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM20 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_21 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM21 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM30 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_32 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM32 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM35 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_37 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM37 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_50 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM50 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_52 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM52 %s
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_53 %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCH64 -check-prefix SM53 %s

// SM20:--image=profile=sm_20{{.*}}--image=profile=compute_20
// SM21:--image=profile=sm_21{{.*}}--image=profile=compute_20
// SM30:--image=profile=sm_30{{.*}}--image=profile=compute_30
// SM32:--image=profile=sm_32{{.*}}--image=profile=compute_32
// SM35:--image=profile=sm_35{{.*}}--image=profile=compute_35
// SM37:--image=profile=sm_37{{.*}}--image=profile=compute_37
// SM50:--image=profile=sm_50{{.*}}--image=profile=compute_50
// SM52:--image=profile=sm_52{{.*}}--image=profile=compute_52
// SM53:--image=profile=sm_53{{.*}}--image=profile=compute_53
