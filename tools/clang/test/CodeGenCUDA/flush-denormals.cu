// RUN: %clang_cc1 -fcuda-is-device \
// RUN:   -triple nvptx-nvidia-cuda -emit-llvm -o - %s | \
// RUN:   FileCheck %s -check-prefix CHECK -check-prefix NOFTZ
// RUN: %clang_cc1 -fcuda-is-device -fcuda-flush-denormals-to-zero \
// RUN:   -triple nvptx-nvidia-cuda -emit-llvm -o - %s | \
// RUN:   FileCheck %s -check-prefix CHECK -check-prefix FTZ

#include "Inputs/hip/hip_runtime.h"

// Checks that device function calls get emitted with the "ntpvx-f32ftz"
// attribute set to "true" when we compile CUDA device code with
// -fcuda-flush-denormals-to-zero.  Further, check that we reflect the presence
// or absence of -fcuda-flush-denormals-to-zero in a module flag.

// CHECK-LABEL: define void @foo() #0
extern "C" __device__ void foo() {}

// FTZ: attributes #0 = {{.*}} "nvptx-f32ftz"="true"
// NOFTZ-NOT: attributes #0 = {{.*}} "nvptx-f32ftz"

// FTZ:!llvm.module.flags = !{{{.*}}[[MODFLAG:![0-9]+]]}
// FTZ:[[MODFLAG]] = !{i32 4, !"nvvm-reflect-ftz", i32 1}

// NOFTZ:!llvm.module.flags = !{{{.*}}[[MODFLAG:![0-9]+]]}
// NOFTZ:[[MODFLAG]] = !{i32 4, !"nvvm-reflect-ftz", i32 0}
