#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -fcuda-include-gpubinary %t -o - | FileCheck %s
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -fcuda-include-gpubinary %t -o -  -DNOGLOBALS \
// RUN:   | FileCheck %s -check-prefix=NOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -o - | FileCheck %s -check-prefix=NOGPUBIN

#include "Inputs/hip/hip_runtime.h"

#ifndef NOGLOBALS
// CHECK-DAG: @device_var = internal global i32
__device__ int device_var;

// CHECK-DAG: @constant_var = internal global i32
__constant__ int constant_var;

// CHECK-DAG: @shared_var = internal global i32
__shared__ int shared_var;

// Make sure host globals don't get internalized...
// CHECK-DAG: @host_var = global i32
int host_var;
// ... and that extern vars remain external.
// CHECK-DAG: @ext_host_var = external global i32
extern int ext_host_var;

// Shadows for external device-side variables are *definitions* of
// those variables.
// CHECK-DAG: @ext_device_var = internal global i32
extern __device__ int ext_device_var;
// CHECK-DAG: @ext_device_var = internal global i32
extern __constant__ int ext_constant_var;

void use_pointers() {
  int *p;
  p = &device_var;
  p = &constant_var;
  p = &shared_var;
  p = &host_var;
  p = &ext_device_var;
  p = &ext_constant_var;
  p = &ext_host_var;
}

// Make sure that all parts of GPU code init/cleanup are there:
// * constant unnamed string with the kernel name
// CHECK: private unnamed_addr constant{{.*}}kernelfunc{{.*}}\00"
// * constant unnamed string with GPU binary
// CHECK: private unnamed_addr constant{{.*GPU binary would be here.*}}\00"
// CHECK-SAME: section ".nv_fatbin", align 8
// * constant struct that wraps GPU binary
// CHECK: @__cuda_fatbin_wrapper = internal constant { i32, i32, i8*, i8* } 
// CHECK-SAME: { i32 1180844977, i32 1, {{.*}}, i8* null }
// CHECK-SAME: section ".nvFatBinSegment"
// * variable to save GPU binary handle after initialization
// CHECK: @__cuda_gpubin_handle = internal global i8** null
// * Make sure our constructor/destructor was added to global ctor/dtor list.
// CHECK: @llvm.global_ctors = appending global {{.*}}@__cuda_module_ctor
// CHECK: @llvm.global_dtors = appending global {{.*}}@__cuda_module_dtor

// Test that we build the correct number of calls to hipSetupArgument followed
// by a call to hipLaunchByPtr.

// CHECK: define{{.*}}kernelfunc
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipLaunchByPtr
__global__ void kernelfunc(int i, int j, int k) {}

// Test that we've built correct kernel launch sequence.
// CHECK: define{{.*}}hostfunc
// CHECK: call{{.*}}hipConfigureCall
// CHECK: call{{.*}}kernelfunc
void hostfunc(void) { kernelfunc<<<1, 1>>>(1, 1, 1); }
#endif

// Test that we've built a function to register kernels and global vars.
// CHECK: define internal void @__cuda_register_globals
// CHECK: call{{.*}}cudaRegisterFunction(i8** %0, {{.*}}kernelfunc
// CHECK-DAG: call{{.*}}cudaRegisterVar(i8** %0, {{.*}}device_var{{.*}}i32 0, i32 4, i32 0, i32 0
// CHECK-DAG: call{{.*}}cudaRegisterVar(i8** %0, {{.*}}constant_var{{.*}}i32 0, i32 4, i32 1, i32 0
// CHECK-DAG: call{{.*}}cudaRegisterVar(i8** %0, {{.*}}ext_device_var{{.*}}i32 1, i32 4, i32 0, i32 0
// CHECK-DAG: call{{.*}}cudaRegisterVar(i8** %0, {{.*}}ext_constant_var{{.*}}i32 1, i32 4, i32 1, i32 0
// CHECK: ret void

// Test that we've built contructor..
// CHECK: define internal void @__cuda_module_ctor
//   .. that calls __cudaRegisterFatBinary(&__cuda_fatbin_wrapper)
// CHECK: call{{.*}}cudaRegisterFatBinary{{.*}}__cuda_fatbin_wrapper
//   .. stores return value in __cuda_gpubin_handle
// CHECK-NEXT: store{{.*}}__cuda_gpubin_handle
//   .. and then calls __cuda_register_globals
// CHECK-NEXT: call void @__cuda_register_globals

// Test that we've created destructor.
// CHECK: define internal void @__cuda_module_dtor
// CHECK: load{{.*}}__cuda_gpubin_handle
// CHECK-NEXT: call void @__cudaUnregisterFatBinary

// There should be no __cuda_register_globals if we have no
// device-side globals, but we still need to register GPU binary.
// Skip GPU binary string first.
// NOGLOBALS: @0 = private unnamed_addr constant{{.*}}
// NOGLOBALS-NOT: define internal void @__cuda_register_globals
// NOGLOBALS: define internal void @__cuda_module_ctor
// NOGLOBALS: call{{.*}}cudaRegisterFatBinary{{.*}}__cuda_fatbin_wrapper
// NOGLOBALS-NOT: call void @__cuda_register_globals
// NOGLOBALS: define internal void @__cuda_module_dtor
// NOGLOBALS: call void @__cudaUnregisterFatBinary

// There should be no constructors/destructors if we have no GPU binary.
// NOGPUBIN-NOT: define internal void @__cuda_register_globals
// NOGPUBIN-NOT: define internal void @__cuda_module_ctor
// NOGPUBIN-NOT: define internal void @__cuda_module_dtor
