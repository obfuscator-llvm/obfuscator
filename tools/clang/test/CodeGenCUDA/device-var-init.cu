#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target

// Make sure we don't allow dynamic initialization for device
// variables, but accept empty constructors allowed by CUDA.

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck %s

#ifdef __clang__
#include "Inputs/hip/hip_runtime.h"
#endif

// Use the types we share with Sema tests.
#include "Inputs/cuda-initializers.h"

__device__ int d_v;
// CHECK: @d_v = addrspace(1) externally_initialized global i32 0,
__shared__ int s_v;
// CHECK: @s_v = addrspace(3) global i32 undef,
__constant__ int c_v;
// CHECK: addrspace(4) externally_initialized global i32 0,

__device__ int d_v_i = 1;
// CHECK: @d_v_i = addrspace(1) externally_initialized global i32 1,

// trivial constructor -- allowed
__device__ T d_t;
// CHECK: @d_t = addrspace(1) externally_initialized global %struct.T zeroinitializer
__shared__ T s_t;
// CHECK: @s_t = addrspace(3) global %struct.T undef,
__constant__ T c_t;
// CHECK: @c_t = addrspace(4) externally_initialized global %struct.T zeroinitializer,

__device__ T d_t_i = {2};
// CHECK: @d_t_i = addrspace(1) externally_initialized global %struct.T { i32 2 },
__constant__ T c_t_i = {2};
// CHECK: @c_t_i = addrspace(4) externally_initialized global %struct.T { i32 2 },

// empty constructor
__device__ EC d_ec;
// CHECK: @d_ec = addrspace(1) externally_initialized global %struct.EC zeroinitializer,
__shared__ EC s_ec;
// CHECK: @s_ec = addrspace(3) global %struct.EC undef,
__constant__ EC c_ec;
// CHECK: @c_ec = addrspace(4) externally_initialized global %struct.EC zeroinitializer,

// empty destructor
__device__ ED d_ed;
// CHECK: @d_ed = addrspace(1) externally_initialized global %struct.ED zeroinitializer,
__shared__ ED s_ed;
// CHECK: @s_ed = addrspace(3) global %struct.ED undef,
__constant__ ED c_ed;
// CHECK: @c_ed = addrspace(4) externally_initialized global %struct.ED zeroinitializer,

__device__ ECD d_ecd;
// CHECK: @d_ecd = addrspace(1) externally_initialized global %struct.ECD zeroinitializer,
__shared__ ECD s_ecd;
// CHECK: @s_ecd = addrspace(3) global %struct.ECD undef,
__constant__ ECD c_ecd;
// CHECK: @c_ecd = addrspace(4) externally_initialized global %struct.ECD zeroinitializer,

// empty templated constructor -- allowed with no arguments
__device__ ETC d_etc;
// CHECK: @d_etc = addrspace(1) externally_initialized global %struct.ETC zeroinitializer,
__shared__ ETC s_etc;
// CHECK: @s_etc = addrspace(3) global %struct.ETC undef,
__constant__ ETC c_etc;
// CHECK: @c_etc = addrspace(4) externally_initialized global %struct.ETC zeroinitializer,

__device__ NCFS d_ncfs;
// CHECK: @d_ncfs = addrspace(1) externally_initialized global %struct.NCFS { i32 3 }
__constant__ NCFS c_ncfs;
// CHECK: @c_ncfs = addrspace(4) externally_initialized global %struct.NCFS { i32 3 }

// Regular base class -- allowed
__device__ T_B_T d_t_b_t;
// CHECK: @d_t_b_t = addrspace(1) externally_initialized global %struct.T_B_T zeroinitializer,
__shared__ T_B_T s_t_b_t;
// CHECK: @s_t_b_t = addrspace(3) global %struct.T_B_T undef,
__constant__ T_B_T c_t_b_t;
// CHECK: @c_t_b_t = addrspace(4) externally_initialized global %struct.T_B_T zeroinitializer,

// Incapsulated object of allowed class -- allowed
__device__ T_F_T d_t_f_t;
// CHECK: @d_t_f_t = addrspace(1) externally_initialized global %struct.T_F_T zeroinitializer,
__shared__ T_F_T s_t_f_t;
// CHECK: @s_t_f_t = addrspace(3) global %struct.T_F_T undef,
__constant__ T_F_T c_t_f_t;
// CHECK: @c_t_f_t = addrspace(4) externally_initialized global %struct.T_F_T zeroinitializer,

// array of allowed objects -- allowed
__device__ T_FA_T d_t_fa_t;
// CHECK: @d_t_fa_t = addrspace(1) externally_initialized global %struct.T_FA_T zeroinitializer,
__shared__ T_FA_T s_t_fa_t;
// CHECK: @s_t_fa_t = addrspace(3) global %struct.T_FA_T undef,
__constant__ T_FA_T c_t_fa_t;
// CHECK: @c_t_fa_t = addrspace(4) externally_initialized global %struct.T_FA_T zeroinitializer,


// Calling empty base class initializer is OK
__device__ EC_I_EC d_ec_i_ec;
// CHECK: @d_ec_i_ec = addrspace(1) externally_initialized global %struct.EC_I_EC zeroinitializer,
__shared__ EC_I_EC s_ec_i_ec;
// CHECK: @s_ec_i_ec = addrspace(3) global %struct.EC_I_EC undef,
__constant__ EC_I_EC c_ec_i_ec;
// CHECK: @c_ec_i_ec = addrspace(4) externally_initialized global %struct.EC_I_EC zeroinitializer,

// We should not emit global initializers for device-side variables.
// CHECK-NOT: @__cxx_global_var_init

// Make sure that initialization restrictions do not apply to local
// variables.
__device__ void df() {
  T t;
  // CHECK-NOT: call
  EC ec;
  // CHECK:   call void @_ZN2ECC1Ev(%struct.EC* %ec)
  ED ed;
  // CHECK-NOT: call
  ECD ecd;
  // CHECK:   call void @_ZN3ECDC1Ev(%struct.ECD* %ecd)
  ETC etc;
  // CHECK:   call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* %etc)
  UC uc;
  // undefined constructor -- not allowed
  // CHECK:   call void @_ZN2UCC1Ev(%struct.UC* %uc)
  UD ud;
  // undefined destructor -- not allowed
  // CHECK-NOT: call
  ECI eci;
  // empty constructor w/ initializer list -- not allowed
  // CHECK:   call void @_ZN3ECIC1Ev(%struct.ECI* %eci)
  NEC nec;
  // non-empty constructor -- not allowed
  // CHECK:   call void @_ZN3NECC1Ev(%struct.NEC* %nec)
  // non-empty destructor -- not allowed
  NED ned;
  // no-constructor,  virtual method -- not allowed
  // CHECK:   call void @_ZN3NCVC1Ev(%struct.NCV* %ncv)
  NCV ncv;
  // CHECK-NOT: call
  VD vd;
  // CHECK:   call void @_ZN2VDC1Ev(%struct.VD* %vd)
  NCF ncf;
  // CHECK:   call void @_ZN3NCFC1Ev(%struct.NCF* %ncf)
  NCFS ncfs;
  // CHECK:   call void @_ZN4NCFSC1Ev(%struct.NCFS* %ncfs)
  UTC utc;
  // CHECK:   call void @_ZN3UTCC1IJEEEDpT_(%struct.UTC* %utc)
  NETC netc;
  // CHECK:   call void @_ZN4NETCC1IJEEEDpT_(%struct.NETC* %netc)
  T_B_T t_b_t;
  // CHECK-NOT: call
  T_F_T t_f_t;
  // CHECK-NOT: call
  T_FA_T t_fa_t;
  // CHECK-NOT: call
  EC_I_EC ec_i_ec;
  // CHECK:   call void @_ZN7EC_I_ECC1Ev(%struct.EC_I_EC* %ec_i_ec)
  EC_I_EC1 ec_i_ec1;
  // CHECK:   call void @_ZN8EC_I_EC1C1Ev(%struct.EC_I_EC1* %ec_i_ec1)
  T_V_T t_v_t;
  // CHECK:   call void @_ZN5T_V_TC1Ev(%struct.T_V_T* %t_v_t)
  T_B_NEC t_b_nec;
  // CHECK:   call void @_ZN7T_B_NECC1Ev(%struct.T_B_NEC* %t_b_nec)
  T_F_NEC t_f_nec;
  // CHECK:   call void @_ZN7T_F_NECC1Ev(%struct.T_F_NEC* %t_f_nec)
  T_FA_NEC t_fa_nec;
  // CHECK:   call void @_ZN8T_FA_NECC1Ev(%struct.T_FA_NEC* %t_fa_nec)
  T_B_NED t_b_ned;
  // CHECK-NOT: call
  T_F_NED t_f_ned;
  // CHECK-NOT: call
  T_FA_NED t_fa_ned;
  // CHECK-NOT: call
  static __shared__ EC s_ec;
  // CHECK-NOT: call void @_ZN2ECC1Ev(%struct.EC* addrspacecast (%struct.EC addrspace(3)* @_ZZ2dfvE4s_ec to %struct.EC*))
  static __shared__ ETC s_etc;
  // CHECK-NOT: call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* addrspacecast (%struct.ETC addrspace(3)* @_ZZ2dfvE5s_etc to %struct.ETC*))

  // anchor point separating constructors and destructors
  df(); // CHECK: call void @_Z2dfv()

  // Verify that we only call non-empty destructors
  // CHECK-NEXT: call void @_ZN8T_FA_NEDD1Ev(%struct.T_FA_NED* %t_fa_ned)
  // CHECK-NEXT: call void @_ZN7T_F_NEDD1Ev(%struct.T_F_NED* %t_f_ned)
  // CHECK-NEXT: call void @_ZN7T_B_NEDD1Ev(%struct.T_B_NED* %t_b_ned)
  // CHECK-NEXT: call void @_ZN2VDD1Ev(%struct.VD* %vd)
  // CHECK-NEXT: call void @_ZN3NEDD1Ev(%struct.NED* %ned)
  // CHECK-NEXT: call void @_ZN2UDD1Ev(%struct.UD* %ud)
  // CHECK-NEXT: call void @_ZN3ECDD1Ev(%struct.ECD* %ecd)
  // CHECK-NEXT: call void @_ZN2EDD1Ev(%struct.ED* %ed)

  // CHECK-NEXT: ret void
}

// We should not emit global init function.
// CHECK-NOT: @_GLOBAL__sub_I
